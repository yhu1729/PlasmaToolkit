#include <hip/hip_runtime.h>

__global__ void
kernel_add(const int n, const double* x, double* y) {
  int index = blockDim.x * blockIdx.x + threadIdx.x;
  if (index < n) {
    y[index] += x[index];
  }
}

extern "C" {
void
invoke_kernel_add(const int n, const double* x, double* y) {
  double* x_d;
  double* y_d;
  hipMalloc((void**)&x_d, n * sizeof(double));
  hipMalloc((void**)&y_d, n * sizeof(double));

  hipMemcpy(x_d, x, n * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(y_d, y, n * sizeof(double), hipMemcpyHostToDevice);

  const int n_thread_per_block = 256;
  const int n_block_per_grid =
    (n + n_thread_per_block - 1) / n_thread_per_block;
  kernel_add<<<n_block_per_grid, n_thread_per_block>>>(n, x_d, y_d);

  hipMemcpy(y, y_d, n * sizeof(double), hipMemcpyDeviceToHost);

  hipFree(x_d);
  hipFree(y_d);
}
}
