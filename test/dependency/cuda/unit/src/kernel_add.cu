#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

__global__ void
kernel_add(const int n, const double* x, double* y) {
  int index = blockDim.x * blockIdx.x + threadIdx.x;
  if (index < n) {
    y[n] = y[n] + x[n];
  }
}

void
invoke_kernel_add(const int n, const double* x_d, const double* y_d) {
  const int n_thread_per_block = 256;
  const int n_block_per_grid =
    (n + n_thread_per_block - 1) / n_thread_per_block;
  kernel_add<<<n_block_per_grid, n_thread_per_block>>>(n, x_d, y_d);
}
